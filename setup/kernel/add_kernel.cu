#include "hip/hip_runtime.h"
#include "my_add.h"

__global__ void add_kernel(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

void add2(float* c, const float* a, const float* b, int n) {
    int block_size = 1024;
    int grid_size = (n + block_size - 1) / block_size;
    add_kernel<<<grid_size, block_size>>>(a, b, c, n);
}
